#include "hip/hip_runtime.h"
#include "common.h"

Ray::Ray(GzCoord aOrigin, GzCoord aDirection){
	for(int i = 0; i < 3; i++){
		origin[i] = aOrigin[i];
		direction[i] = aDirection[i];
	}
}

Triangle::Triangle(GzCoord aVertexList[], GzCoord aNormalList[], GzTextureIndex aUVList[]){
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < 3; j++){
			vertexList[i][j] = aVertexList[i][j];
			normalList[i][j] = aNormalList[i][j];
			if(j != 2){
				uvList[i][j] = aUVList[i][j];
			}
		}
	}
}

void Triangle::setVertexList(GzCoord aVertexList[]){
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < 3; j++){
			vertexList[i][j] = aVertexList[i][j];
		}
	}
}

void Triangle::setNormalList(GzCoord aNormalList[]){
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < 3; j++){
			normalList[i][j] = aNormalList[i][j];
		}
	}
}

void Triangle::setUVList(GzTextureIndex aUVList[]){
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < 2; j++){
			uvList[i][j] = aUVList[i][j];
		}
	}
}

Plane::Plane(GzCoord aNormal, float aDistance){
	for(int i = 0; i < 3; i++) normal[i] = aNormal[i];
	distance = aDistance;
}

Plane::Plane(GzCoord aVertexList[]){
	GzCoord temp1, temp2;
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < 3; j++){
			vertexList[i][j] = aVertexList[i][j];
		}
	}

	vectorConstruct(vertexList[0], vertexList[1], temp1);
	vectorConstruct(vertexList[0], vertexList[2], temp2);
	vectorCrossProduct(temp1, temp2, normal);
	vectorNormalize(normal);

	distance = vectorDotProduct(normal, vertexList[0]);
}

int Plane::findIntersectPoint(Ray& aRay, GzCoord aPoint){

	GzCoord w0, temp;

	if(vectorZero(normal)) return -1; //triangle degenerate to a point

	vectorConstruct(vertexList[0], aRay.origin, w0);
	float a = -vectorDotProduct(normal, w0);
	float b = vectorDotProduct(normal, aRay.direction);

	if(fabs(b) < SMALL_NUM){ //ray is parallel to triangle
		if(a == 0){
			return 2; //ray lies in triangle plane
		}else{
			return 0; //ray disjoint from triangle plane
		}
	}

	float r = a/b;
	if(r < 0.0) return 0; //ray goes away from triangle

	vectorScale(r, aRay.direction, temp);
	vectorAdd(aRay.origin, temp, aPoint);

	return 1; //one intersect point 
}

bool Plane::checkPointInTriangle(GzCoord aPoint){
	GzCoord vector0, vector1, vector2;

	//compute vectors
	vectorConstruct(vertexList[0], vertexList[2], vector0);
	vectorConstruct(vertexList[0], vertexList[1], vector1);
	vectorConstruct(vertexList[0], aPoint, vector2);

	//compute dot products
	float dot00 = vectorDotProduct(vector0, vector0);
	float dot01 = vectorDotProduct(vector0, vector1);
	float dot02 = vectorDotProduct(vector0, vector2);
	float dot11 = vectorDotProduct(vector1, vector1);
	float dot12 = vectorDotProduct(vector1, vector2);

	//compute barycentric coordinates
	float invDenom = 1.0/(dot00*dot11 - dot01*dot01);
	float u = (dot11*dot02 - dot01*dot12) * invDenom;
	float v = (dot00*dot12 - dot01*dot02) * invDenom;

	//check if point is in the triangle
	return (u >= 0) && (v >= 0) && (u + v < 1.0);
}
