#include "hip/hip_runtime.h"
/* CS580 Homework 3 */

#include	"stdafx.h"
#include	"stdio.h"
#include	"math.h"
#include	"Gz.h"
#include	"rend.h"
#include	"book.h"
#include	"hip/hip_runtime.h"

#include <limits.h>

__host__ __device__ short ctoi(float color);
__host__ __device__ bool vectorZero(GzCoord vector);
__host__ __device__ void vectorConstruct(GzCoord vector1, GzCoord vector2, GzCoord vector); //vector = vector2 - vector1
__host__ __device__ void vectorNormalize(GzCoord vector);
__host__ __device__ float vectorDotProduct(GzCoord vector1, GzCoord vector2); //return vector1 * vector2
__host__ __device__ void vectorCrossProduct(GzCoord vector1, GzCoord vector2, GzCoord product); //product = vector1 x vector2
__host__ __device__ void vectorScale(float scale, GzCoord vector);
__host__ __device__ void vectorScale(float scale, GzCoord vector1, GzCoord vector2);

__host__ __device__ float vectorTransform(GzCoord vector, float w, GzMatrix matrix);
__host__ __device__ void matrixMultiply(GzMatrix matrixA, GzMatrix matrixB, GzMatrix matrixC);

__host__ __device__ void vectorAdd(GzCoord vector1, GzCoord vector2);
__host__ __device__ void vectorAdd(GzCoord vector1, GzCoord vector2, GzCoord vector3);
__host__ __device__ void vectorMultiply(GzCoord vector1, GzCoord vector2);
__host__ __device__ void vectorMultiply(GzCoord vector1, GzCoord vector2, GzCoord vector3);
__host__ __device__ float vectorLength(GzCoord vector);

int GzRotXMat(float degree, GzMatrix mat)
{
// Create rotate matrix : rotate along x axis
// Pass back the matrix using mat value

	mat[0][0] = 1.0;
	mat[0][1] = 0.0;
	mat[0][2] = 0.0;
	mat[0][3] = 0.0;

	mat[1][0] = 0.0;
	mat[1][1] = cos(PIII/180*degree);
	mat[1][2] = -sin(PIII/180*degree);
	mat[1][3] = 0.0;

	mat[2][0] = 0.0;
	mat[2][1] = sin(PIII/180*degree);
	mat[2][2] = cos(PIII/180*degree);
	mat[2][3] = 0.0;

	mat[3][0] = 0.0;
	mat[3][1] = 0.0;
	mat[3][2] = 0.0;
	mat[3][3] = 1.0;
	
	return GZ_SUCCESS;
}


int GzRotYMat(float degree, GzMatrix mat)
{
// Create rotate matrix : rotate along y axis
// Pass back the matrix using mat value

	mat[0][0] = cos(PIII/180*degree);
	mat[0][1] = 0.0;
	mat[0][2] = sin(PIII/180*degree);
	mat[0][3] = 0.0;

	mat[1][0] = 0.0;
	mat[1][1] = 1.0;
	mat[1][2] = 0.0;
	mat[1][3] = 0.0;

	mat[2][0] = -sin(PIII/180*degree);
	mat[2][1] = 0.0;
	mat[2][2] = cos(PIII/180*degree);
	mat[2][3] = 0.0;

	mat[3][0] = 0.0;
	mat[3][1] = 0.0;
	mat[3][2] = 0.0;
	mat[3][3] = 1.0;

	return GZ_SUCCESS;
}


int GzRotZMat(float degree, GzMatrix mat)
{
// Create rotate matrix : rotate along z axis
// Pass back the matrix using mat value

	mat[0][0] = cos(PIII/180*degree);
	mat[0][1] = -sin(PIII/180*degree);
	mat[0][2] = 0.0;
	mat[0][3] = 0.0;

	mat[1][0] = sin(PIII/180*degree);
	mat[1][1] = cos(PIII/180*degree);
	mat[1][2] = 0.0;
	mat[1][3] = 0.0;

	mat[2][0] = 0.0;
	mat[2][1] = 0.0;
	mat[2][2] = 1.0;
	mat[2][3] = 0.0;

	mat[3][0] = 0.0;
	mat[3][1] = 0.0;
	mat[3][2] = 0.0;
	mat[3][3] = 1.0;

	return GZ_SUCCESS;
}


int GzTrxMat(GzCoord translate, GzMatrix mat)
{
// Create translation matrix
// Pass back the matrix using mat value

	mat[0][0] = 1.0;
	mat[0][1] = 0.0;
	mat[0][2] = 0.0;
	mat[0][3] = translate[0];

	mat[1][0] = 0.0;
	mat[1][1] = 1.0;
	mat[1][2] = 0.0;
	mat[1][3] = translate[1];

	mat[2][0] = 0.0;
	mat[2][1] = 0.0;
	mat[2][2] = 1.0;
	mat[2][3] = translate[2];

	mat[3][0] = 0.0;
	mat[3][1] = 0.0;
	mat[3][2] = 0.0;
	mat[3][3] = 1.0;

	return GZ_SUCCESS;
}


int GzScaleMat(GzCoord scale, GzMatrix mat)
{
// Create scaling matrix
// Pass back the matrix using mat value

	mat[0][0] = scale[0];
	mat[0][1] = 0.0;
	mat[0][2] = 0.0;
	mat[0][3] = 0.0;

	mat[1][0] = 0.0;
	mat[1][1] = scale[1];
	mat[1][2] = 0.0;
	mat[1][3] = 0.0;

	mat[2][0] = 0.0;
	mat[2][1] = 0.0;
	mat[2][2] = scale[2];
	mat[2][3] = 0.0;

	mat[3][0] = 0.0;
	mat[3][1] = 0.0;
	mat[3][2] = 0.0;
	mat[3][3] = 1.0;

	return GZ_SUCCESS;
}


//----------------------------------------------------------
// Begin main functions

int GzNewRender(GzRender **render, GzRenderClass renderClass, GzDisplay	*display)
{
/*  
- malloc a renderer struct 
- keep closed until all inits are done 
- setup Xsp and anything only done once 
- span interpolator needs pointer to display 
- check for legal class GZ_Z_BUFFER_RENDER 
- init default camera 
*/ 
	GzRender* oneRender = NULL;

	//malloc a new renderer struct
	oneRender = (GzRender*)malloc(sizeof(GzRender));

	if(oneRender == NULL || renderClass != GZ_Z_BUFFER_RENDER){
		return GZ_FAILURE;
	}else{
		//keep render close until all inits are done
		oneRender->renderClass = renderClass;
		oneRender->display = display;
		oneRender->matlevel = -1; //matrix index is initialized to -1
		oneRender->numlights = 0; //light index is inialized to 0

		oneRender->triangleBuffer = new vector<Triangle>();
		oneRender->numTriangle = 0;

		//setup Xsp;
		oneRender->Xsp[0][0] = display->xres/2.0;
		oneRender->Xsp[0][1] = 0.0;
		oneRender->Xsp[0][2] = 0.0;
		oneRender->Xsp[0][3] = display->xres/2.0;

		oneRender->Xsp[1][0] = 0.0;
		oneRender->Xsp[1][1] = -display->yres/2.0;
		oneRender->Xsp[1][2] = 0.0;
		oneRender->Xsp[1][3] = display->yres/2.0;

		oneRender->Xsp[2][0] = 0.0;
		oneRender->Xsp[2][1] = 0.0;
		oneRender->Xsp[2][2] = tan((PIII/180)*DEFAULT_FOV/2.0) * INT_MAX;
		oneRender->Xsp[2][3] = 0.0;

		oneRender->Xsp[3][0] = 0.0;
		oneRender->Xsp[3][1] = 0.0;
		oneRender->Xsp[3][2] = 0.0;
		oneRender->Xsp[3][3] = 1.0;

		//init default camera
		oneRender->camera.FOV = DEFAULT_FOV;
		oneRender->camera.lookat[0] = 0.0;
		oneRender->camera.lookat[1] = 0.0;
		oneRender->camera.lookat[2] = 0.0;
		oneRender->camera.position[0] = DEFAULT_IM_X;
		oneRender->camera.position[1] = DEFAULT_IM_Y;
		oneRender->camera.position[2] = DEFAULT_IM_Z;
		oneRender->camera.worldup[0] = 0.0;
		oneRender->camera.worldup[1] = 1.0;
		oneRender->camera.worldup[2] = 0.0;

		oneRender->open = 1;
	}

	*render = oneRender;

	return GZ_SUCCESS;

}


int GzFreeRender(GzRender *render)
{
/* 
-free all renderer resources
*/
	if(render == NULL){
		return GZ_FAILURE;
	}else{
		free(render);
	}

	return GZ_SUCCESS;
}


int GzBeginRender(GzRender *render)
{
/*  
- set up for start of each frame - clear frame buffer 
- compute Xiw and projection xform Xpi from camera definition 
- init Ximage - put Xsp at base of stack, push on Xpi and Xiw 
- now stack contains Xsw and app can push model Xforms if it want to. 
*/ 

	unsigned short x, y;
	GzDisplay* display;

	float xCoord[3];
	float yCoord[3];
	float zCoord[3];

	if(render == NULL){
		return GZ_FAILURE;
	}else{	
		display = render->display;
	}

	//clear frame buffer
	for(y = 0; y < display->yres; y++){
		for(x = 0; x < display->xres; x++){
			display->fbuf[ARRAY(x,y)].red = 0x0fff;
			display->fbuf[ARRAY(x,y)].green = 0x0fff;
			display->fbuf[ARRAY(x,y)].blue = 0x0fff;
			display->fbuf[ARRAY(x,y)].alpha = 1;
			display->fbuf[ARRAY(x,y)].z = 0;
		}
	}

	render->Xsp[0][0] = display->xres/2.0;
	render->Xsp[0][1] = 0.0;
	render->Xsp[0][2] = 0.0;
	render->Xsp[0][3] = display->xres/2.0;

	render->Xsp[1][0] = 0.0;
	render->Xsp[1][1] = -display->yres/2.0;
	render->Xsp[1][2] = 0.0;
	render->Xsp[1][3] = display->yres/2.0;

	render->Xsp[2][0] = 0.0;
	render->Xsp[2][1] = 0.0;
	render->Xsp[2][2] = tan((PIII/180)*render->camera.FOV/2.0) * INT_MAX;
	render->Xsp[2][3] = 0.0;

	render->Xsp[3][0] = 0.0;
	render->Xsp[3][1] = 0.0;
	render->Xsp[3][2] = 0.0;
	render->Xsp[3][3] = 1.0;

	//compute Xpi
	render->camera.Xpi[0][0] = 1.0;
	render->camera.Xpi[0][1] = 0.0;
	render->camera.Xpi[0][2] = 0.0;
	render->camera.Xpi[0][3] = 0.0;

	render->camera.Xpi[1][0] = 0.0;
	render->camera.Xpi[1][1] = 1.0;
	render->camera.Xpi[1][2] = 0.0;
	render->camera.Xpi[1][3] = 0.0;

	render->camera.Xpi[2][0] = 0.0;
	render->camera.Xpi[2][1] = 0.0;
	render->camera.Xpi[2][2] = 1.0;
	render->camera.Xpi[2][3] = 0.0;

	render->camera.Xpi[3][0] = 0.0;
	render->camera.Xpi[3][1] = 0.0;
	render->camera.Xpi[3][2] = tan((PIII/180)*render->camera.FOV/2.0);
	render->camera.Xpi[3][3] = 1.0;

	//compute Xiw
	vectorConstruct(render->camera.position, render->camera.lookat, zCoord);
	vectorNormalize(zCoord);
	float temp = vectorDotProduct(render->camera.worldup, zCoord);
	vectorScale(temp, zCoord);
	vectorConstruct(zCoord, render->camera.worldup, yCoord); //correct one
	vectorNormalize(yCoord);
	vectorScale(1/temp, zCoord);
	vectorCrossProduct(yCoord, zCoord, xCoord);

	render->camera.Xiw[0][0] = xCoord[0];
	render->camera.Xiw[0][1] = xCoord[1];
	render->camera.Xiw[0][2] = xCoord[2];
	render->camera.Xiw[0][3] = -1 * vectorDotProduct(xCoord, render->camera.position);

	render->camera.Xiw[1][0] = yCoord[0];
	render->camera.Xiw[1][1] = yCoord[1];
	render->camera.Xiw[1][2] = yCoord[2];
	render->camera.Xiw[1][3] = -1 * vectorDotProduct(yCoord, render->camera.position);

	render->camera.Xiw[2][0] = zCoord[0];
	render->camera.Xiw[2][1] = zCoord[1];
	render->camera.Xiw[2][2] = zCoord[2];
	render->camera.Xiw[2][3] = -1 *vectorDotProduct(zCoord, render->camera.position);

	render->camera.Xiw[3][0] = 0.0;
	render->camera.Xiw[3][1] = 0.0;
	render->camera.Xiw[3][2] = 0.0;
	render->camera.Xiw[3][3] = 1.0;

	//init stack
	GzPushMatrix(render, render->Xsp);
	GzPushMatrix(render, render->camera.Xpi);
	GzPushMatrix(render, render->camera.Xiw);

	return GZ_SUCCESS;
}

int GzPutCamera(GzRender *render, GzCamera *camera)
{
/*
- overwrite renderer camera structure with new camera definition
*/
	int i, j;

	if(render == NULL || camera == NULL){
		return GZ_FAILURE;
	}
/*
	for(i = 0; i < 4; i++){
		for(j = 0; j < 4; j++){
			render->camera.Xiw[i][j] = camera->Xiw[i][j];
			render->camera.Xpi[i][j] = camera->Xpi[i][j];
		}
	}
*/
	for(i = 0; i < 3; i++){
		render->camera.lookat[i] = camera->lookat[i];
		render->camera.position[i] = camera->position[i];
		render->camera.worldup[i] = camera->worldup[i];
	}

	render->camera.FOV = camera->FOV;
	return GZ_SUCCESS;	
}

int GzPushMatrix(GzRender *render, GzMatrix	matrix)
{
/*
- push a matrix onto the Ximage stack
- check for stack overflow
*/
	int i, j;
	GzMatrix dummy = {
		1.0, 0.0, 0.0, 0.0,
		0.0, 1.0, 0.0, 0.0,
		0.0, 0.0, 1.0, 0.0,
		0.0, 0.0, 0.0, 1.0
	};

	GzMatrix temp = {
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0
	};
	float k;

	if(render->matlevel+1 == MATLEVELS){
		return GZ_FAILURE;
	}

	(render->matlevel)++;

	if(render->matlevel == 0){
		for(i = 0; i < 4; i++){
			for(j = 0; j < 4; j++){
				render->Ximage[render->matlevel][i][j] = matrix[i][j];
				render->Xnorm[render->matlevel][i][j] = dummy[i][j];
				render->Xraytrace[render->matlevel][i][j] = dummy[i][j];
			}
		}
	}else{
		matrixMultiply(render->Ximage[(render->matlevel)-1], matrix, render->Ximage[render->matlevel]);
		if(render->matlevel == 1){
			matrixMultiply(render->Xnorm[(render->matlevel)-1], dummy, render->Xnorm[render->matlevel]);
			matrixMultiply(render->Xraytrace[(render->matlevel)-1], dummy, render->Xraytrace[render->matlevel]);
		}else{
			for(i = 0; i < 4; i++){
				for(j = 0; j < 4; j++){
					temp[i][j] = matrix[i][j];
				}
			}
			//remove the translation
			temp[0][3] = 0.0;
			temp[1][3] = 0.0;
			temp[2][3] = 0.0;

			//normalize the rotation
			k = 1/sqrt(temp[0][0]*temp[0][0] + temp[0][1]*temp[0][1] + temp[0][2]*temp[0][2] + temp[0][3]*temp[0][3]);
			for(i = 0; i < 3; i++){
				for(j = 0; j < 3; j++){
					temp[i][j] = temp[i][j] * k;
				}
			}

			//push matrix into Xnorm stack
			matrixMultiply(render->Xnorm[(render->matlevel)-1], temp, render->Xnorm[render->matlevel]);
			matrixMultiply(render->Xraytrace[(render->matlevel)-1], matrix, render->Xraytrace[render->matlevel]);
		}
	}

	return GZ_SUCCESS;
}

int GzPopMatrix(GzRender *render)
{
/*
- pop a matrix off the Ximage stack
- check for stack underflow
*/
	if(render->matlevel-1 == -1){
		return GZ_FAILURE;
	}

	render->matlevel--;

	return GZ_SUCCESS;
}

void copyDirectionalLightParam(GzRender *render, int index, GzLight *light){
	for(int i = 0; i < 3; i++){
		render->lights[index].direction[i] = light->direction[i];
		render->lights[index].color[i] = light->color[i];
	}
}

void copyAmbientLightParam(GzRender *render, GzLight *light){
	for(int i = 0; i < 3; i++){
		render->ambientlight.direction[i] = light->direction[i];
		render->ambientlight.color[i] = light->color[i];
	}
}

void copyCoefficient(GzRender *render, int type, float *coef){
	for(int i = 0; i < 3; i++){
		if(type == GZ_DIFFUSE_COEFFICIENT){
			render->Kd[i] = coef[i];
		}else if(type == GZ_AMBIENT_COEFFICIENT){
			render->Ka[i] = coef[i];
		}else if(type == GZ_SPECULAR_COEFFICIENT){
			render->Ks[i] = coef[i];
		}
	}
}

int GzPutAttribute(GzRender	*render, int numAttributes, GzToken	*nameList, 
	GzPointer	*valueList) /* void** valuelist */
{
/*
- set renderer attribute states (e.g.: GZ_RGB_COLOR default color)
- later set shaders, interpolaters, texture maps, and lights
*/
	if(render == NULL || nameList == NULL || valueList == NULL){
		return GZ_FAILURE;
	}

	for(int i = 0; i < numAttributes; i++){
		if(nameList[i] == GZ_RGB_COLOR){
			render->flatcolor[0] = *((float*)valueList[i]);
			render->flatcolor[1] = *((float*)valueList[i] + 1);
			render->flatcolor[2] = *((float*)valueList[i] + 2);
		}else if(nameList[i] == GZ_DIRECTIONAL_LIGHT){
			copyDirectionalLightParam(render, render->numlights, (GzLight*)valueList[i]);
			render->numlights++;
		}else if(nameList[i] == GZ_AMBIENT_LIGHT){
			copyAmbientLightParam(render, (GzLight*)valueList[i]);
		}else if(nameList[i] == GZ_DIFFUSE_COEFFICIENT){
			copyCoefficient(render, GZ_DIFFUSE_COEFFICIENT, (float*)valueList[i]);
		}else if(nameList[i] == GZ_AMBIENT_COEFFICIENT){
			copyCoefficient(render, GZ_AMBIENT_COEFFICIENT, (float*)valueList[i]);
		}else if(nameList[i] == GZ_SPECULAR_COEFFICIENT){
			copyCoefficient(render, GZ_SPECULAR_COEFFICIENT, (float*)valueList[i]);
		}else if(nameList[i] == GZ_DISTRIBUTION_COEFFICIENT){
			render->spec = *((float*)valueList[i]);
		}else if(nameList[i] == GZ_INTERPOLATE){
			if(*((int*)valueList[i]) == GZ_COLOR){
				render->interp_mode = GZ_COLOR;
			}else if(*((int*)valueList[i]) == GZ_NORMALS){
				render->interp_mode = GZ_NORMALS;
			}else if(*((int*)valueList[i]) == GZ_FLAT){
				render->interp_mode = GZ_FLAT;
			}
		}else if(nameList[i] == GZ_TEXTURE_MAP){
			render->tex_fun = (GzTexture)valueList[i];
		}
	}

	return GZ_SUCCESS;
}

void bubbleSort(float* vertexArray[], int size){

	bool swap = true;
	int j = 0;
	float* temp;
	
	while(swap){
		swap = false;
		j++;
		for(int i = 0; i < size-j; i++){
			if(vertexArray[i][1] > vertexArray[i+1][1]){
				temp = vertexArray[i];
				vertexArray[i] = vertexArray[i+1];
				vertexArray[i+1] = temp;
				swap = true;
			}
		}
	}

	//handle the case of top edge
	if(vertexArray[0][1] == vertexArray[1][1]){
		if(vertexArray[0][0] > vertexArray[1][0]){
			temp = vertexArray[0];
			vertexArray[0] = vertexArray[1];
			vertexArray[1] = temp;
		}
	}

	//handle the case of bottom edge
	if(vertexArray[1][1] == vertexArray[2][1]){
		if(vertexArray[1][0] > vertexArray[2][0]){
			temp = vertexArray[1];
			vertexArray[1] = vertexArray[2];
			vertexArray[2] = temp;
		}
	}
}

void bubbleSort(float* vertexArray[], int size, float* colorArray[], float* normalArray[], float* uvArray[]){

	bool swap = true;
	int j = 0;
	float* temp;
	
	while(swap){
		swap = false;
		j++;
		for(int i = 0; i < size-j; i++){
			if(vertexArray[i][1] > vertexArray[i+1][1]){
				//flip vertex array
				temp = vertexArray[i];
				vertexArray[i] = vertexArray[i+1];
				vertexArray[i+1] = temp;

				//flip color ptr array
				temp = colorArray[i];
				colorArray[i] = colorArray[i+1];
				colorArray[i+1] = temp;

				//flip normal array
				temp = normalArray[i];
				normalArray[i] = normalArray[i+1];
				normalArray[i+1] = temp;

				//flip UV array
				temp = uvArray[i];
				uvArray[i] = uvArray[i+1];
				uvArray[i+1] = temp;

				swap = true;
			}
		}
	}

	//handle the case of top edge
	if(vertexArray[0][1] == vertexArray[1][1]){
		if(vertexArray[0][0] > vertexArray[1][0]){
			temp = vertexArray[0];
			vertexArray[0] = vertexArray[1];
			vertexArray[1] = temp;
		}
	}

	//handle the case of bottom edge
	if(vertexArray[1][1] == vertexArray[2][1]){
		if(vertexArray[1][0] > vertexArray[2][0]){
			temp = vertexArray[1];
			vertexArray[1] = vertexArray[2];
			vertexArray[2] = temp;
		}
	}
}

int GzSaveTriangle(GzRender *render, int numParts, GzToken *nameList, GzPointer *valueList){

	int i, j;
	float* vertexArray[3];
	float* normalArray[3];
	float* uvArray[3];

	Triangle* aTriangle = new Triangle();

	for(i = 0; i < numParts; i++){
		if(nameList[i] == GZ_POSITION){
			for(j = 0; j < 3; j++){
				vertexArray[j] = (float*)valueList[i] + j*3;
				vectorTransform(vertexArray[j], 1.0, render->Xraytrace[render->matlevel]);
			}
			aTriangle->setVertexList((GzCoord*)valueList[i]);
		}else if(nameList[i] == GZ_NORMAL){
			for(j = 0; j < 3; j++){
				normalArray[j] = (float*)valueList[i] + j*3;
				vectorTransform(normalArray[j], 1.0, render->Xnorm[render->matlevel]);
			}
			aTriangle->setNormalList((GzCoord*)valueList[i]);
		}else if(nameList[i] == GZ_TEXTURE_INDEX){
			aTriangle->setUVList((GzTextureIndex*)valueList[i]);
		}
	}

	render->triangleBuffer->push_back(*aTriangle);
	render->numTriangle = render->numTriangle + 1;

	return GZ_SUCCESS;
}

__host__ __device__ float triangleArea(GzCoord vertexA, GzCoord vertexB, GzCoord vertexC){

	GzCoord vectorBA = {0.0, 0.0, 0.0};
	GzCoord vectorCA = {0.0, 0.0, 0.0};
	GzCoord product = {0.0, 0.0, 0.0};

	vectorConstruct(vertexA, vertexB, vectorBA);
	vectorConstruct(vertexA, vertexC, vectorCA);
	vectorCrossProduct(vectorBA, vectorCA, product);

	return vectorLength(product)/2.0;
}

__host__ __device__ float vertexDistance(GzCoord vertexA, GzCoord vertexB){

	GzCoord tmp = {0.0, 0.0, 0.0};
	vectorConstruct(vertexA, vertexB, tmp);
	return vectorLength(tmp);
}

__device__ void GzPutFrameBuffer(GzPixel* fbuf, int offset, 
								GzIntensity r, GzIntensity g, GzIntensity b, GzIntensity a, GzDepth z){

	if(fbuf[offset].z == 0 || z < fbuf[offset].z){

		//clamp the red color
		if(r > 4095){
			fbuf[offset].red = 4095;
		}else if(r < 0){
			fbuf[offset].red = 0;
		}else{
			fbuf[offset].red = r;
		}

		//clamp the green color
		if(g > 4095){
			fbuf[offset].green = 4095;
		}else if(g < 0){
			fbuf[offset].green = 0;
		}else{
			fbuf[offset].green = g;
		}

		//clamp the blue color
		if(b > 4095){
			fbuf[offset].blue = 4095;
		}else if(b < 0){
			fbuf[offset].blue = 0;
		}else{
			fbuf[offset].blue = b;
		}

		fbuf[offset].alpha = a;
		fbuf[offset].z = z;
	}
}


__device__ int findIntersectPoint(GzCoord origin, GzCoord direction, GzCoord normal, GzCoord* vertexList, GzCoord aPoint){

	GzCoord w0, temp;

	if(vectorZero(normal)) return -1; //triangle degenerate to a point

	vectorConstruct(vertexList[0], origin, w0);
	float a = -vectorDotProduct(normal, w0);
	float b = vectorDotProduct(normal, direction);

	if(fabs(b) < SMALL_NUM){ //ray is parallel to triangle
		if(a == 0){
			return 2; //ray lies in triangle plane
		}else{
			return 0; //ray disjoint from triangle plane
		}
	}

	float r = a/b;
	if(r < 0.0) return 0; //ray goes away from triangle

	vectorScale(r, direction, temp);
	vectorAdd(origin, temp, aPoint);

	return 1; //one intersect point 
}

__device__ bool checkPointInTriangle(GzCoord* vertexList, GzCoord aPoint){
	GzCoord vector0, vector1, vector2;

	//compute vectors
	vectorConstruct(vertexList[0], vertexList[2], vector0);
	vectorConstruct(vertexList[0], vertexList[1], vector1);
	vectorConstruct(vertexList[0], aPoint, vector2);

	//compute dot products
	float dot00 = vectorDotProduct(vector0, vector0);
	float dot01 = vectorDotProduct(vector0, vector1);
	float dot02 = vectorDotProduct(vector0, vector2);
	float dot11 = vectorDotProduct(vector1, vector1);
	float dot12 = vectorDotProduct(vector1, vector2);

	//compute barycentric coordinates
	float invDenom = 1.0/(dot00*dot11 - dot01*dot01);
	float u = (dot11*dot02 - dot01*dot12) * invDenom;
	float v = (dot00*dot12 - dot01*dot02) * invDenom;

	//check if point is in the triangle
	return (u >= 0) && (v >= 0) && (u + v < 1.0);
}

__global__ void kernel(TriangleCUDA* triBuffer, GzPixel* frameBuffer, GzRenderCUDA* render, float originZ){

	int x, y, i, j, k, it, offset;
	GzCoord origin = {0.0, 0.0, 0.0};
	GzCoord screenPoint = {0.0, 0.0, 0.0};
	GzCoord rayDirection = {0.0, 0.0, 0.0};
	GzCoord intersection = {0.0, 0.0, 0.0};
	GzCoord interNormal = {0.0, 0.0, 0.0};

	GzCoord eVector = {0.0, 0.0, -1.0};
	GzCoord rVector = {0.0, 0.0, 0.0};
	GzCoord tempVector1 = {0.0, 0.0, 0.0};
	GzCoord tempVector2 = {0.0, 0.0, 0.0};
	GzCoord tempVector3 = {0.0, 0.0, 0.0};
	GzCoord sumVector1 = {0.0, 0.0, 0.0};
	GzCoord sumVector2 = {0.0, 0.0, 0.0};
	GzCoord color = {0.0, 0.0, 0.0};
	float resultNL, resultNE, resultRE, result;

	//plane
	GzCoord normal = {0.0, 0.0, 0.0};
	GzCoord temp1 = {0.0, 0.0, 0.0};
	GzCoord temp2 = {0.0, 0.0, 0.0};
	float distance;
	
	origin[2] = originZ;
	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;
	offset = y * blockDim.x * gridDim.x + x;

	screenPoint[0] = (float)x/256.0 - 0.5;
	screenPoint[1] = (float)y/256.0 - 0.5; 
	vectorConstruct(origin, screenPoint, rayDirection);
	vectorNormalize(rayDirection);
			
	for(k = 0; k < render->numTriangle; k++){

		vectorConstruct(triBuffer[k].vertexList[0], triBuffer[k].vertexList[1], temp1);
		vectorConstruct(triBuffer[k].vertexList[0], triBuffer[k].vertexList[2], temp2);
		vectorCrossProduct(temp1, temp2, normal);
		vectorNormalize(normal);
		distance = vectorDotProduct(normal, triBuffer[k].vertexList[0]);
				
		if(findIntersectPoint(origin, rayDirection, normal, triBuffer[k].vertexList, intersection) == 1){
			if(checkPointInTriangle(triBuffer[k].vertexList, intersection)){

				//interpalate the normal at intersection point
				float areaTotal = triangleArea(triBuffer[k].vertexList[0], triBuffer[k].vertexList[1], triBuffer[k].vertexList[2]);
				float area01 = triangleArea(intersection, triBuffer[k].vertexList[0], triBuffer[k].vertexList[1]);
				float area02 = triangleArea(intersection, triBuffer[k].vertexList[0], triBuffer[k].vertexList[2]);
				float area12 = triangleArea(intersection, triBuffer[k].vertexList[1], triBuffer[k].vertexList[2]);

				interNormal[0] = (area12/areaTotal)*triBuffer[k].normalList[0][0] + 
					(area02/areaTotal)*triBuffer[k].normalList[1][0] + (area01/areaTotal)*triBuffer[k].normalList[2][0];
				interNormal[1] = (area12/areaTotal)*triBuffer[k].normalList[0][1] + 
					(area02/areaTotal)*triBuffer[k].normalList[1][1] + (area01/areaTotal)*triBuffer[k].normalList[2][1];
				interNormal[2] = (area12/areaTotal)*triBuffer[k].normalList[0][2] +
					(area02/areaTotal)*triBuffer[k].normalList[1][2] + (area01/areaTotal)*triBuffer[k].normalList[2][2];
						
						
				//calculate the shading at intersection point using Phone shading

				for(it = 0; it < render->numlights; it++){
					resultNL = vectorDotProduct(interNormal, render->lights[it].direction);
					resultNE = vectorDotProduct(interNormal, eVector);

					if(resultNL * resultNE > 0){
						if(resultNL < 0 && resultNE < 0){
							//flip normal
							interNormal[0] = -interNormal[0];
							interNormal[1] = -interNormal[1];
							interNormal[2] = -interNormal[2];
							resultNL = vectorDotProduct(interNormal, render->lights[it].direction);
							resultNE = vectorDotProduct(interNormal, eVector);
						}

						vectorScale(2*resultNL, interNormal);
						vectorConstruct(render->lights[it].direction, interNormal, rVector);
						vectorScale(1/(2*resultNL), interNormal);
						resultRE = vectorDotProduct(rVector, eVector) < 0 ? 0 : vectorDotProduct(rVector, eVector);
						result = pow(resultRE, render->spec);
						vectorScale(result, render->lights[it].color, sumVector1);
						vectorAdd(tempVector1, sumVector1);
				
						vectorScale(resultNL, render->lights[it].color, sumVector2);
						vectorAdd(tempVector2, sumVector2);
					}
				}

				vectorMultiply(tempVector1, render->Ks);
				vectorMultiply(tempVector2, render->Kd);
				vectorMultiply(render->ambientlight.color, render->Ka, tempVector3);

				vectorAdd(color, tempVector1);
				vectorAdd(color, tempVector2);
				vectorAdd(color, tempVector3);

				for(it = 0; it < 3; it++){
					if(color[it] > 1.0) color[it] = 1.0;
				}

				//update the color of pixel in frame buffer
				GzPutFrameBuffer(frameBuffer, offset, ctoi(color[0]), ctoi(color[1]), ctoi(color[2]),
					1, vertexDistance(origin, intersection));
			}
		}
	}

	return;	

}

int GzRayTraceCUDA(GzRender *render){

	TriangleCUDA* triBuffer_host;
	TriangleCUDA* triBuffer_device;

	GzPixel* frameBuffer_device;

	GzRenderCUDA* render_host;
	GzRenderCUDA* render_device;

	Triangle aTriangle;

	hipEvent_t start, stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	float origin = -1.0/tan((PIII/180)*render->camera.FOV/2.0);

	triBuffer_host = (TriangleCUDA*)malloc(render->numTriangle*sizeof(TriangleCUDA));
	render_host = (GzRenderCUDA*)malloc(sizeof(GzRenderCUDA));

	HANDLE_ERROR(hipMalloc((void**)&triBuffer_device, render->numTriangle*sizeof(TriangleCUDA)));
	HANDLE_ERROR(hipMalloc((void**)&frameBuffer_device, (render->display->xres)*(render->display->yres)*sizeof(GzPixel)));
	HANDLE_ERROR(hipMalloc((void**)&render_device, sizeof(GzRenderCUDA)));

	for(int i = 0; i < render->numTriangle; i++){
		aTriangle = render->triangleBuffer->at(i);
		for(int j = 0; j < 3; j++){
			for(int k = 0; k < 3; k++){
				triBuffer_host[i].vertexList[j][k] = aTriangle.vertexList[j][k];
				triBuffer_host[i].normalList[j][k] = aTriangle.normalList[j][k];
				if(k != 2){
					triBuffer_host[i].uvList[j][k] = aTriangle.uvList[j][k];
				}
			}
		}
	}

	render_host->numlights = render->numlights;
	render_host->spec = render->spec;
	render_host->numTriangle = render->numTriangle;
	for(int i = 0; i < render->numlights; i++){
		for(int j = 0; j < 3; j++){
			render_host->lights[i].direction[j] = render->lights[i].direction[j];
			render_host->lights[i].color[j] = render->lights[i].color[j];
		}
	}
	for(int i = 0; i < 3; i++){
		render_host->ambientlight.direction[i] = render->ambientlight.direction[i];
		render_host->ambientlight.color[i] = render->ambientlight.color[i];
		render_host->Ka[i] = render->Ka[i];
		render_host->Kd[i] = render->Kd[i];
		render_host->Ks[i] = render->Ks[i];
	}

	HANDLE_ERROR(hipMemcpy(triBuffer_device, triBuffer_host, render->numTriangle*sizeof(TriangleCUDA), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(render_device, render_host, sizeof(GzRenderCUDA), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(frameBuffer_device, render->display->fbuf, (render->display->xres)*(render->display->yres)*sizeof(GzPixel),
		hipMemcpyHostToDevice));

	dim3 grids(render->display->xres/DIM, render->display->yres/DIM);
	dim3 blocks(DIM, DIM);

	kernel<<<grids, blocks>>>(triBuffer_device, frameBuffer_device, render_device, origin);

	HANDLE_ERROR(hipMemcpy(render->display->fbuf, frameBuffer_device, 
	(render->display->xres)*(render->display->yres)*sizeof(GzPixel), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float time;
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
	printf("Time to render: %3.1f ms\n", time);
	
	HANDLE_ERROR(hipFree(triBuffer_device));
	HANDLE_ERROR(hipFree(frameBuffer_device));
	HANDLE_ERROR(hipFree(render_device));
	
	free(triBuffer_host);
	free(render_host);

	return GZ_SUCCESS;
}

int GzRayTraceRender(GzRender *render){

	int i, j, k, it;
	GzCoord origin = {0.0, 0.0, -1.0/tan((PIII/180)*render->camera.FOV/2.0)};
	GzCoord screenPoint = {0.0, 0.0, 0.0};
	GzCoord rayDirection = {0.0, 0.0, 0.0};
	GzCoord intersection = {0.0, 0.0, 0.0};
	GzCoord interNormal = {0.0, 0.0, 0.0};

	GzCoord eVector = {0.0, 0.0, -1.0};
	GzCoord rVector = {0.0, 0.0, 0.0};
	GzCoord tempVector1 = {0.0, 0.0, 0.0};
	GzCoord tempVector2 = {0.0, 0.0, 0.0};
	GzCoord tempVector3 = {0.0, 0.0, 0.0};
	GzCoord sumVector1 = {0.0, 0.0, 0.0};
	GzCoord sumVector2 = {0.0, 0.0, 0.0};
	GzCoord color = {0.0, 0.0, 0.0};
	float resultNL, resultNE, resultRE, result;

	for(i = 0; i < render->display->yres; i++){
		for(j = 0; j < render->display->xres; j++){
			screenPoint[0] = (float)j/(float)(render->display->xres) - 0.5;
			screenPoint[1] = (float)i/(float)(render->display->yres) - 0.5; 
			vectorConstruct(origin, screenPoint, rayDirection);
			vectorNormalize(rayDirection);

			Ray* aRay = new Ray(origin, rayDirection);
			
			for(k = 0; k < render->triangleBuffer->size(); k++){
				Triangle aTriangle = render->triangleBuffer->at(k);
				Plane* aPlane = new Plane(aTriangle.vertexList);
				
				if(aPlane->findIntersectPoint(*aRay, intersection) == 1){
					if(aPlane->checkPointInTriangle(intersection)){

						//interpalate the normal at intersection point
						float areaTotal = triangleArea(aPlane->vertexList[0], aPlane->vertexList[1], aPlane->vertexList[2]);
						float area01 = triangleArea(intersection, aPlane->vertexList[0], aPlane->vertexList[1]);
						float area02 = triangleArea(intersection, aPlane->vertexList[0], aPlane->vertexList[2]);
						float area12 = triangleArea(intersection, aPlane->vertexList[1], aPlane->vertexList[2]);

						interNormal[0] = (area12/areaTotal)*aTriangle.normalList[0][0] + 
							(area02/areaTotal)*aTriangle.normalList[1][0] + (area01/areaTotal)*aTriangle.normalList[2][0];
						interNormal[1] = (area12/areaTotal)*aTriangle.normalList[0][1] + 
							(area02/areaTotal)*aTriangle.normalList[1][1] + (area01/areaTotal)*aTriangle.normalList[2][1];
						interNormal[2] = (area12/areaTotal)*aTriangle.normalList[0][2] +
							(area02/areaTotal)*aTriangle.normalList[1][2] + (area01/areaTotal)*aTriangle.normalList[2][2];
						
						
						//calculate the shading at intersection point using Phone shading
						for(it = 0; it < 3; it++){
							tempVector1[it] = 0.0;
							tempVector2[it] = 0.0;
							color[it] = 0.0;
						}

						for(it = 0; it < render->numlights; it++){
							resultNL = vectorDotProduct(interNormal, render->lights[it].direction);
							resultNE = vectorDotProduct(interNormal, eVector);

							if(resultNL * resultNE > 0){
								if(resultNL < 0 && resultNE < 0){
									//flip normal
									interNormal[0] = -interNormal[0];
									interNormal[1] = -interNormal[1];
									interNormal[2] = -interNormal[2];
									resultNL = vectorDotProduct(interNormal, render->lights[it].direction);
									resultNE = vectorDotProduct(interNormal, eVector);
								}

								vectorScale(2*resultNL, interNormal);
								vectorConstruct(render->lights[it].direction, interNormal, rVector);
								vectorScale(1/(2*resultNL), interNormal);
								resultRE = vectorDotProduct(rVector, eVector) < 0 ? 0 : vectorDotProduct(rVector, eVector);
								result = pow(resultRE, render->spec);
								vectorScale(result, render->lights[it].color, sumVector1);
								vectorAdd(tempVector1, sumVector1);
				
								vectorScale(resultNL, render->lights[it].color, sumVector2);
								vectorAdd(tempVector2, sumVector2);
							}
						}

						vectorMultiply(tempVector1, render->Ks);
						vectorMultiply(tempVector2, render->Kd);
						vectorMultiply(render->ambientlight.color, render->Ka, tempVector3);

						vectorAdd(color, tempVector1);
						vectorAdd(color, tempVector2);
						vectorAdd(color, tempVector3);

						for(it = 0; it < 3; it++){
							if(color[it] > 1.0) color[it] = 1.0;
						}

						//update the color of pixel in frame buffer
						GzPutDisplay(render->display, j, i, ctoi(color[0]), ctoi(color[1]), ctoi(color[2]),
							1, vertexDistance(aRay->origin, intersection));
					}
				}

				delete aPlane;
			}

			delete aRay;
		}
	}

	return GZ_SUCCESS;
}

int GzPutTriangle(GzRender	*render, int numParts, GzToken *nameList, 
				  GzPointer	*valueList)
/* numParts : how many names and values */
{
/*  
- pass in a triangle description with tokens and values corresponding to 
      GZ_POSITION:3 vert positions in model space 
- Xform positions of verts  
- Clip - just discard any triangle with verts behind view plane 
       - test for triangles with all three verts off-screen 
- invoke triangle rasterizer  
*/
	float* vertexArray[3];
	GzDDA ddaArray[3];
	GzDDA* leftEdge;
	GzDDA* rightEdge;
	GzDDA spanDDA;

	int i, j, k, x, y;
	float deltaY, deltaX, w;
	GzDisplay* display;
	float deltaY23;

	//shading variables
	float* normalArray[3];
	float colorArray[3][3] = {
		0.0, 0.0, 0.0,
		0.0, 0.0, 0.0,
		0.0, 0.0, 0.0
	};
	GzColor pixelColor = {0.0, 0.0, 0.0};

	GzCoord eVector = {0.0, 0.0, -1.0};
	GzCoord rVector = {0.0, 0.0, 0.0};
	GzCoord tempVector1 = {0.0, 0.0, 0.0};
	GzCoord tempVector2 = {0.0, 0.0, 0.0};
	GzCoord tempVector3 = {0.0, 0.0, 0.0};
	GzCoord sumVector1 = {0.0, 0.0, 0.0};
	GzCoord sumVector2 = {0.0, 0.0, 0.0};
	float resultNL, resultNE, resultRE, result;
	float* colorPtrArray[3];

	//texturing variables
	float* uvArray[3];
	GzColor kColor = {0.0, 0.0, 0.0};
	float uTemp, vTemp, vzPrime1, vzPrime2;

	for(i = 0; i < 3; i++) colorPtrArray[i] = &colorArray[i][0];
	if(render == NULL || nameList == NULL || valueList == NULL){
		return GZ_FAILURE;
	}
	display = render->display;

	if(render->interp_mode == GZ_FLAT){
		for(i = 0; i < numParts; i++){
			if(nameList[i] != GZ_NULL_TOKEN){
				if(nameList[i] == GZ_POSITION){

					//inialize the vertexArray
					for(j = 0; j < 3; j++){
						vertexArray[j] = (float*)valueList[i] + j*3;
					}

					for(j = 0; j < 3; j++){
						w = 1.0;
						w = vectorTransform(vertexArray[j], w, render->Ximage[render->matlevel]);
						if(vertexArray[j][2] < 0){
							return GZ_SUCCESS; //ignore the triangle which has vertex behind the camera
						}
						vertexArray[j][0] = vertexArray[j][0]/w;
						vertexArray[j][1] = vertexArray[j][1]/w;
						vertexArray[j][2] = vertexArray[j][2]/w;
					}

					//invoke the scan line rasterizer
					//sort the vertices based on Y coordinates
					bubbleSort(vertexArray, 3);
				
					//initialize the DDA0 for 1-2 edge
					for(j = 0; j < 3; j++){
						ddaArray[0].start[j] = vertexArray[0][j];
						ddaArray[0].end[j] = vertexArray[1][j];
						ddaArray[0].current[j] = vertexArray[0][j];
					}
					if(vertexArray[1][1] != vertexArray[0][1]){
						ddaArray[0].slopeX = (vertexArray[1][0]-vertexArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeZ = (vertexArray[1][2]-vertexArray[0][2])/(vertexArray[1][1]-vertexArray[0][1]);
					}

					//inialize the DDA1 for 1-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[1].start[j] = vertexArray[0][j];
						ddaArray[1].end[j] = vertexArray[2][j];
						ddaArray[1].current[j] = vertexArray[0][j];
					}
					ddaArray[1].slopeX = (vertexArray[2][0]-vertexArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeZ = (vertexArray[2][2]-vertexArray[0][2])/(vertexArray[2][1]-vertexArray[0][1]);

					//inialize the DDA2 for 2-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[2].start[j] = vertexArray[1][j];
						ddaArray[2].end[j] = vertexArray[2][j];
						ddaArray[2].current[j] = vertexArray[1][j];
					}
					if(vertexArray[2][1] != vertexArray[1][1]){
						ddaArray[2].slopeX = (vertexArray[2][0]-vertexArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeZ = (vertexArray[2][2]-vertexArray[1][2])/(vertexArray[2][1]-vertexArray[1][1]);
					}

					deltaY = ceil(vertexArray[0][1]) - vertexArray[0][1];
					deltaY23 = ceil(vertexArray[1][1]) - vertexArray[1][1];
					if(vertexArray[1][1] != vertexArray[0][1]){
						//assign left or right edges
						if(ddaArray[0].slopeX < ddaArray[1].slopeX){
							leftEdge = &(ddaArray[0]);
							rightEdge = &(ddaArray[1]);
						}else{
							leftEdge = &(ddaArray[1]);
							rightEdge = &(ddaArray[0]);
						}

						//advance 1-2 DDA0 current position to top ceilling position
						ddaArray[0].current[0] = ddaArray[0].current[0] + ddaArray[0].slopeX * deltaY;
						ddaArray[0].current[1] = ddaArray[0].current[1] + deltaY;
						ddaArray[0].current[2] = ddaArray[0].current[2] + ddaArray[0].slopeZ * deltaY;

						//advance 1-3 DDA1 current position to top ceilling position
						ddaArray[1].current[0] = ddaArray[1].current[0] + ddaArray[1].slopeX * deltaY;
						ddaArray[1].current[1] = ddaArray[1].current[1] + deltaY;
						ddaArray[1].current[2] = ddaArray[1].current[2] + ddaArray[1].slopeZ * deltaY;

						//advance 2-3 DDA2 current position to top ceilling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY23;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY23;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY23;
					}else{
						leftEdge = &(ddaArray[1]);
						rightEdge = &(ddaArray[2]);

						//advance 2-3 DDA2 current position to top ceiling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY;
					}		

					while(ddaArray[1].current[1] <= vertexArray[2][1] && ddaArray[1].current[1] < 257 ){

						//switch from 1-2 edge to 2-3 edge
						if(ddaArray[0].current[1] >= vertexArray[1][1] && vertexArray[1][1] != vertexArray[0][1]){
							if(ddaArray[0].slopeX < ddaArray[1].slopeX){
								leftEdge = &(ddaArray[2]);
							}else{
								rightEdge = &(ddaArray[2]);
							}
						}

						//inialize the span DDA
						for(j = 0; j < 3; j++){
							spanDDA.start[j] = leftEdge->current[j];
							spanDDA.end[j] = rightEdge->current[j];
							spanDDA.current[j] = leftEdge->current[j];
						}

						spanDDA.slopeX = 0.0f;
						spanDDA.slopeZ = (rightEdge->current[2]-leftEdge->current[2])/(rightEdge->current[0]-leftEdge->current[0]);

						//advance span DDA current position to left-most covered pixel
						deltaX = ceil(leftEdge->current[0]) - leftEdge->current[0];
						spanDDA.current[0] = spanDDA.current[0] + deltaX;
						spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ * deltaX;

						while(spanDDA.current[0] <= spanDDA.end[0]){
							//write color value into the frame buffer
							x = spanDDA.current[0];
							y = spanDDA.current[1];
							GzPutDisplay(render->display, x, y, ctoi(render->flatcolor[0]), ctoi(render->flatcolor[1]), ctoi(render->flatcolor[2]),
									1, spanDDA.current[2]);

							//update span DDA current position
							(spanDDA.current[0])++;
							spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ;
						}

						//update left and right edge current position
						leftEdge->current[0] = leftEdge->current[0] + leftEdge->slopeX;
						(leftEdge->current[1])++;
						leftEdge->current[2] = leftEdge->current[2] + leftEdge->slopeZ;

						rightEdge->current[0] = rightEdge->current[0] + rightEdge->slopeX;
						(rightEdge->current[1])++;
						rightEdge->current[2] = rightEdge->current[2] + rightEdge->slopeZ;
					}
				}
			}
		}
	}else{
		//shading code 

		for(i = 0; i < numParts; i++){
			if(nameList[i] == GZ_POSITION){
				//initialize the vertexArray
				for(j = 0; j < 3; j++){
					vertexArray[j] = (float*)valueList[i] + j*3;
				}

			}else if(nameList[i] == GZ_NORMAL){
				//initialize the normalArray
				for(j = 0; j < 3; j++){
					normalArray[j] = (float*)valueList[i] + j*3;
				}
			}else if(nameList[i] == GZ_TEXTURE_INDEX){
				//initialize the uvArray
				for(j = 0; j < 3; j++){
					uvArray[j] = (float*)valueList[i] + j*2;
				}
			}
		}

		//transform vertex with Ximage
		for(j = 0; j < 3; j++){
			w = 1.0;
			w = vectorTransform(vertexArray[j], w, render->Ximage[render->matlevel]);
/*			if(vertexArray[j][2] < 0){
				return GZ_SUCCESS; //ignore the triangle which has vertex behind the camera
			}
*/
			vertexArray[j][0] = vertexArray[j][0]/w;
			vertexArray[j][1] = vertexArray[j][1]/w;
			vertexArray[j][2] = vertexArray[j][2]/w;

			//warp process: transform uv of vertex to UV
			vzPrime1 = vertexArray[j][2]/((float)INT_MAX - vertexArray[j][2]);
			uvArray[j][0] = uvArray[j][0]/(vzPrime1 + 1.0);
			uvArray[j][1] = uvArray[j][1]/(vzPrime1 + 1.0);

		}

		//transform normal with Xn
		for(j = 0; j < 3; j++){
			w = 1.0;
			w = vectorTransform(normalArray[j], w, render->Xnorm[render->matlevel]);
		}

		if(render->interp_mode == GZ_COLOR){ //for Gauraud shading
			//compute color at each vertex
			for(i = 0; i < 3; i++){
				for(j = 0; j < 3; j++){
					tempVector1[j] = 0.0;
					tempVector2[j] = 0.0;
				}
				for(j = 0; j < render->numlights; j++){
					resultNL = vectorDotProduct(normalArray[i], render->lights[j].direction);
					resultNE = vectorDotProduct(normalArray[i], eVector);

					if(resultNL * resultNE > 0){
						if(resultNL < 0 && resultNE < 0){
							//flip normal
							normalArray[i][0] = -normalArray[i][0];
							normalArray[i][1] = -normalArray[i][1];
							normalArray[i][2] = -normalArray[i][2];
							resultNL = vectorDotProduct(normalArray[i], render->lights[j].direction);
							resultNE = vectorDotProduct(normalArray[i], eVector);
						}

						vectorScale(2*resultNL, normalArray[i]);
						vectorConstruct(render->lights[j].direction, normalArray[i], rVector);
						vectorScale(1/(2*resultNL), normalArray[i]);
						resultRE = vectorDotProduct(rVector, eVector) < 0 ? 0 : vectorDotProduct(rVector, eVector);
						result = pow(resultRE, render->spec);
						vectorScale(result, render->lights[j].color, sumVector1);
						vectorAdd(tempVector1, sumVector1);
				
						vectorScale(resultNL, render->lights[j].color, sumVector2);
						vectorAdd(tempVector2, sumVector2);
					}
				}

//				vectorMultiply(tempVector1, render->Ks);
//				vectorMultiply(tempVector2, render->Kd);
//				vectorMultiply(render->ambientlight.color, render->Ka, tempVector3);

				vectorAdd(colorArray[i], tempVector1);
				vectorAdd(colorArray[i], tempVector2);
				vectorAdd(colorArray[i], render->ambientlight.color);

				for(j = 0; j < 3; j++){
					if(colorArray[i][j] > 1.0) colorArray[i][j] = 1.0;
				}
			}

			
			//rasterize while interpolating RGB color
					//invoke the scan line rasterizer
					//sort the vertices based on Y coordinates
			bubbleSort(vertexArray, 3, colorPtrArray, normalArray, uvArray);
				
					//initialize the DDA0 for 1-2 edge
					for(j = 0; j < 3; j++){
						ddaArray[0].start[j] = vertexArray[0][j];
						ddaArray[0].end[j] = vertexArray[1][j];
						ddaArray[0].current[j] = vertexArray[0][j];

						ddaArray[0].startColor[j] = colorPtrArray[0][j];
						ddaArray[0].endColor[j] = colorPtrArray[1][j];
						ddaArray[0].currentColor[j] = colorPtrArray[0][j];

						if(j != 2){
							ddaArray[0].startUV[j] = uvArray[0][j];
							ddaArray[0].endUV[j] = uvArray[1][j];
							ddaArray[0].currentUV[j] = uvArray[0][j];
						}
					}
					if(vertexArray[1][1] != vertexArray[0][1]){
						ddaArray[0].slopeX = (vertexArray[1][0]-vertexArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeZ = (vertexArray[1][2]-vertexArray[0][2])/(vertexArray[1][1]-vertexArray[0][1]);

						ddaArray[0].slopeR = (colorPtrArray[1][0]-colorPtrArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeG = (colorPtrArray[1][1]-colorPtrArray[0][1])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeB = (colorPtrArray[1][2]-colorPtrArray[0][2])/(vertexArray[1][1]-vertexArray[0][1]);

						ddaArray[0].slopeU = (uvArray[1][0]-uvArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeV = (uvArray[1][1]-uvArray[0][1])/(vertexArray[1][1]-vertexArray[0][1]);
					}

					//inialize the DDA1 for 1-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[1].start[j] = vertexArray[0][j];
						ddaArray[1].end[j] = vertexArray[2][j];
						ddaArray[1].current[j] = vertexArray[0][j];

						ddaArray[1].startColor[j] = colorPtrArray[0][j];
						ddaArray[1].endColor[j] = colorPtrArray[2][j];
						ddaArray[1].currentColor[j] = colorPtrArray[0][j];

						if(j != 2){
							ddaArray[1].startUV[j] = uvArray[0][j];
							ddaArray[1].endUV[j] = uvArray[2][j];
							ddaArray[1].currentUV[j] = uvArray[0][j];
						}
					}
					ddaArray[1].slopeX = (vertexArray[2][0]-vertexArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeZ = (vertexArray[2][2]-vertexArray[0][2])/(vertexArray[2][1]-vertexArray[0][1]);

					ddaArray[1].slopeR = (colorPtrArray[2][0]-colorPtrArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeG = (colorPtrArray[2][1]-colorPtrArray[0][1])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeB = (colorPtrArray[2][2]-colorPtrArray[0][2])/(vertexArray[2][1]-vertexArray[0][1]);

					ddaArray[1].slopeU = (uvArray[2][0]-uvArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeV = (uvArray[2][1]-uvArray[0][1])/(vertexArray[2][1]-vertexArray[0][1]);

					//inialize the DDA2 for 2-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[2].start[j] = vertexArray[1][j];
						ddaArray[2].end[j] = vertexArray[2][j];
						ddaArray[2].current[j] = vertexArray[1][j];

						ddaArray[2].startColor[j] = colorPtrArray[1][j];
						ddaArray[2].endColor[j] = colorPtrArray[2][j];
						ddaArray[2].currentColor[j] = colorPtrArray[1][j];

						if(j != 2){
							ddaArray[2].startUV[j] = uvArray[1][j];
							ddaArray[2].endUV[j] = uvArray[2][j];
							ddaArray[2].currentUV[j] = uvArray[1][j];
						}
					}
					if(vertexArray[2][1] != vertexArray[1][1]){
						ddaArray[2].slopeX = (vertexArray[2][0]-vertexArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeZ = (vertexArray[2][2]-vertexArray[1][2])/(vertexArray[2][1]-vertexArray[1][1]);

						ddaArray[2].slopeR = (colorPtrArray[2][0]-colorPtrArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeG = (colorPtrArray[2][1]-colorPtrArray[1][1])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeB = (colorPtrArray[2][2]-colorPtrArray[1][2])/(vertexArray[2][1]-vertexArray[1][1]);

						ddaArray[2].slopeU = (uvArray[2][0]-uvArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeV = (uvArray[2][1]-uvArray[1][1])/(vertexArray[2][1]-vertexArray[1][1]);
					}

					deltaY = ceil(vertexArray[0][1]) - vertexArray[0][1];
					deltaY23 = ceil(vertexArray[1][1]) - vertexArray[1][1];
					if(vertexArray[1][1] != vertexArray[0][1]){
						//assign left or right edges
						if(ddaArray[0].slopeX < ddaArray[1].slopeX){
							leftEdge = &(ddaArray[0]);
							rightEdge = &(ddaArray[1]);
						}else{
							leftEdge = &(ddaArray[1]);
							rightEdge = &(ddaArray[0]);
						}

						//advance 1-2 DDA0 current position to top ceilling position
						ddaArray[0].current[0] = ddaArray[0].current[0] + ddaArray[0].slopeX * deltaY;
						ddaArray[0].current[1] = ddaArray[0].current[1] + deltaY;
						ddaArray[0].current[2] = ddaArray[0].current[2] + ddaArray[0].slopeZ * deltaY;

						ddaArray[0].currentColor[0] = ddaArray[0].currentColor[0] + ddaArray[0].slopeR * deltaY;
						ddaArray[0].currentColor[1] = ddaArray[0].currentColor[1] + ddaArray[0].slopeG * deltaY;
						ddaArray[0].currentColor[2] = ddaArray[0].currentColor[2] + ddaArray[0].slopeB * deltaY;

						ddaArray[0].currentUV[0] = ddaArray[0].currentUV[0] + ddaArray[0].slopeU * deltaY;
						ddaArray[0].currentUV[1] = ddaArray[0].currentUV[1] + ddaArray[0].slopeV * deltaY;

						//advance 1-3 DDA1 current position to top ceilling position
						ddaArray[1].current[0] = ddaArray[1].current[0] + ddaArray[1].slopeX * deltaY;
						ddaArray[1].current[1] = ddaArray[1].current[1] + deltaY;
						ddaArray[1].current[2] = ddaArray[1].current[2] + ddaArray[1].slopeZ * deltaY;

						ddaArray[1].currentColor[0] = ddaArray[1].currentColor[0] + ddaArray[1].slopeR * deltaY;
						ddaArray[1].currentColor[1] = ddaArray[1].currentColor[1] + ddaArray[1].slopeG * deltaY;
						ddaArray[1].currentColor[2] = ddaArray[1].currentColor[2] + ddaArray[1].slopeB * deltaY;

						ddaArray[1].currentUV[0] = ddaArray[1].currentUV[0] + ddaArray[1].slopeU * deltaY;
						ddaArray[1].currentUV[1] = ddaArray[1].currentUV[1] + ddaArray[1].slopeV * deltaY;

						//advance 2-3 DDA2 current position to top ceilling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY23;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY23;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY23;

						ddaArray[2].currentColor[0] = ddaArray[2].currentColor[0] + ddaArray[2].slopeR * deltaY23;
						ddaArray[2].currentColor[1] = ddaArray[2].currentColor[1] + ddaArray[2].slopeG * deltaY23;
						ddaArray[2].currentColor[2] = ddaArray[2].currentColor[2] + ddaArray[2].slopeB * deltaY23;

						ddaArray[2].currentUV[0] = ddaArray[2].currentUV[0] + ddaArray[2].slopeU * deltaY23;
						ddaArray[2].currentUV[1] = ddaArray[2].currentUV[1] + ddaArray[2].slopeV * deltaY23;
					}else{
						leftEdge = &(ddaArray[1]);
						rightEdge = &(ddaArray[2]);

						//advance 2-3 DDA2 current position to top ceiling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY;
					}		

					while(ddaArray[1].current[1] <= vertexArray[2][1] && ddaArray[1].current[1] < 257 ){

						//switch from 1-2 edge to 2-3 edge
						if(ddaArray[0].current[1] >= vertexArray[1][1] && vertexArray[1][1] != vertexArray[0][1]){
							if(ddaArray[0].slopeX < ddaArray[1].slopeX){
								leftEdge = &(ddaArray[2]);
							}else{
								rightEdge = &(ddaArray[2]);
							}
						}

						//inialize the span DDA
						for(j = 0; j < 3; j++){
							spanDDA.start[j] = leftEdge->current[j];
							spanDDA.end[j] = rightEdge->current[j];
							spanDDA.current[j] = leftEdge->current[j];

							spanDDA.startColor[j] = leftEdge->currentColor[j];
							spanDDA.endColor[j] = rightEdge->currentColor[j];
							spanDDA.currentColor[j] = leftEdge->currentColor[j];

							if(j != 2){
								spanDDA.startUV[j] = leftEdge->currentUV[j];
								spanDDA.endUV[j] = rightEdge->currentUV[j];
								spanDDA.currentUV[j] = leftEdge->currentUV[j];
							}
						}

						spanDDA.slopeX = 0.0f;
						spanDDA.slopeZ = (rightEdge->current[2]-leftEdge->current[2])/(rightEdge->current[0]-leftEdge->current[0]);

						spanDDA.slopeR = (rightEdge->currentColor[0]-leftEdge->currentColor[0])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeG = (rightEdge->currentColor[1]-leftEdge->currentColor[1])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeB = (rightEdge->currentColor[2]-leftEdge->currentColor[2])/(rightEdge->current[0]-leftEdge->current[0]);

						spanDDA.slopeU = (rightEdge->currentUV[0]-leftEdge->currentUV[0])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeV = (rightEdge->currentUV[1]-leftEdge->currentUV[1])/(rightEdge->current[0]-leftEdge->current[0]);

						//advance span DDA current position to left-most covered pixel
						deltaX = ceil(leftEdge->current[0]) - leftEdge->current[0];
						spanDDA.current[0] = spanDDA.current[0] + deltaX;
						spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ * deltaX;

						spanDDA.currentColor[0] = spanDDA.currentColor[0] + spanDDA.slopeR * deltaX;
						spanDDA.currentColor[1] = spanDDA.currentColor[1] + spanDDA.slopeG * deltaX;
						spanDDA.currentColor[2] = spanDDA.currentColor[2] + spanDDA.slopeB * deltaX;

						spanDDA.currentUV[0] = spanDDA.currentUV[0] + spanDDA.slopeU * deltaX;
						spanDDA.currentUV[1] = spanDDA.currentUV[1] + spanDDA.slopeV * deltaX;

						while(spanDDA.current[0] <= spanDDA.end[0]){

							//Gauraud shading unwarp process
							float vzPrime2 = spanDDA.current[2]/((float)INT_MAX - spanDDA.current[2]);
							render->tex_fun(spanDDA.currentUV[0]*(vzPrime2 + 1.0), spanDDA.currentUV[1]*(vzPrime2 + 1.0), kColor);
							vectorMultiply(kColor, spanDDA.currentColor);

							//write color value into the frame buffer
							x = spanDDA.current[0];
							y = spanDDA.current[1];
							GzPutDisplay(render->display, x, y, ctoi(kColor[0]), ctoi(kColor[1]), ctoi(kColor[2]),
									1, spanDDA.current[2]);

							//update span DDA current position
							(spanDDA.current[0])++;
							spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ;

							spanDDA.currentColor[0] = spanDDA.currentColor[0] + spanDDA.slopeR;
							spanDDA.currentColor[1] = spanDDA.currentColor[1] + spanDDA.slopeG;
							spanDDA.currentColor[2] = spanDDA.currentColor[2] + spanDDA.slopeB;

							spanDDA.currentUV[0] = spanDDA.currentUV[0] + spanDDA.slopeU;
							spanDDA.currentUV[1] = spanDDA.currentUV[1] + spanDDA.slopeV;
						}

						//update left and right edge current position
						leftEdge->current[0] = leftEdge->current[0] + leftEdge->slopeX;
						(leftEdge->current[1])++;
						leftEdge->current[2] = leftEdge->current[2] + leftEdge->slopeZ;

						leftEdge->currentColor[0] = leftEdge->currentColor[0] + leftEdge->slopeR;
						leftEdge->currentColor[1] = leftEdge->currentColor[1] + leftEdge->slopeG;
						leftEdge->currentColor[2] = leftEdge->currentColor[2] + leftEdge->slopeB;

						leftEdge->currentUV[0] = leftEdge->currentUV[0] + leftEdge->slopeU;
						leftEdge->currentUV[1] = leftEdge->currentUV[1] + leftEdge->slopeV;

						rightEdge->current[0] = rightEdge->current[0] + rightEdge->slopeX;
						(rightEdge->current[1])++;
						rightEdge->current[2] = rightEdge->current[2] + rightEdge->slopeZ;

						rightEdge->currentColor[0] = rightEdge->currentColor[0] + rightEdge->slopeR;
						rightEdge->currentColor[1] = rightEdge->currentColor[1] + rightEdge->slopeG;
						rightEdge->currentColor[2] = rightEdge->currentColor[2] + rightEdge->slopeB;

						rightEdge->currentUV[0] = rightEdge->currentUV[0] + rightEdge->slopeU;
						rightEdge->currentUV[1] = rightEdge->currentUV[1] + rightEdge->slopeV;
					}

		}else if(render->interp_mode == GZ_NORMALS){ //for Phong shading
			//rasterize while interpolating normals
					bubbleSort(vertexArray, 3, colorPtrArray, normalArray, uvArray);
				
					//initialize the DDA0 for 1-2 edge
					for(j = 0; j < 3; j++){
						ddaArray[0].start[j] = vertexArray[0][j];
						ddaArray[0].end[j] = vertexArray[1][j];
						ddaArray[0].current[j] = vertexArray[0][j];

						ddaArray[0].startNormal[j] = normalArray[0][j];
						ddaArray[0].endNormal[j] = normalArray[1][j];
						ddaArray[0].currentNormal[j] = normalArray[0][j];

						if(j != 2){
							ddaArray[0].startUV[j] = uvArray[0][j];
							ddaArray[0].endUV[j] = uvArray[1][j];
							ddaArray[0].currentUV[j] = uvArray[0][j];
						}
					}
					if(vertexArray[1][1] != vertexArray[0][1]){
						ddaArray[0].slopeX = (vertexArray[1][0]-vertexArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeZ = (vertexArray[1][2]-vertexArray[0][2])/(vertexArray[1][1]-vertexArray[0][1]);

						ddaArray[0].slopeNX = (normalArray[1][0]-normalArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeNY = (normalArray[1][1]-normalArray[0][1])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeNZ = (normalArray[1][2]-normalArray[0][2])/(vertexArray[1][1]-vertexArray[0][1]);

						ddaArray[0].slopeU = (uvArray[1][0]-uvArray[0][0])/(vertexArray[1][1]-vertexArray[0][1]);
						ddaArray[0].slopeV = (uvArray[1][1]-uvArray[0][1])/(vertexArray[1][1]-vertexArray[0][1]);
					}

					//inialize the DDA1 for 1-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[1].start[j] = vertexArray[0][j];
						ddaArray[1].end[j] = vertexArray[2][j];
						ddaArray[1].current[j] = vertexArray[0][j];

						ddaArray[1].startNormal[j] = normalArray[0][j];
						ddaArray[1].endNormal[j] = normalArray[2][j];
						ddaArray[1].currentNormal[j] = normalArray[0][j];

						if(j != 2){
							ddaArray[1].startUV[j] = uvArray[0][j];
							ddaArray[1].endUV[j] = uvArray[2][j];
							ddaArray[1].currentUV[j] = uvArray[0][j];
						}
					}
					ddaArray[1].slopeX = (vertexArray[2][0]-vertexArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeZ = (vertexArray[2][2]-vertexArray[0][2])/(vertexArray[2][1]-vertexArray[0][1]);

					ddaArray[1].slopeNX = (normalArray[2][0]-normalArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeNY = (normalArray[2][1]-normalArray[0][1])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeNZ = (normalArray[2][2]-normalArray[0][2])/(vertexArray[2][1]-vertexArray[0][1]);

					ddaArray[1].slopeU = (uvArray[2][0]-uvArray[0][0])/(vertexArray[2][1]-vertexArray[0][1]);
					ddaArray[1].slopeV = (uvArray[2][1]-uvArray[0][1])/(vertexArray[2][1]-vertexArray[0][1]);

					//inialize the DDA2 for 2-3 edge
					for(j = 0; j < 3; j++){
						ddaArray[2].start[j] = vertexArray[1][j];
						ddaArray[2].end[j] = vertexArray[2][j];
						ddaArray[2].current[j] = vertexArray[1][j];

						ddaArray[2].startNormal[j] = normalArray[1][j];
						ddaArray[2].endNormal[j] = normalArray[2][j];
						ddaArray[2].currentNormal[j] = normalArray[1][j];

						if(j != 2){
							ddaArray[2].startUV[j] = uvArray[1][j];
							ddaArray[2].endUV[j] = uvArray[2][j];
							ddaArray[2].currentUV[j] = uvArray[1][j];
						}
					}
					if(vertexArray[2][1] != vertexArray[1][1]){
						ddaArray[2].slopeX = (vertexArray[2][0]-vertexArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeZ = (vertexArray[2][2]-vertexArray[1][2])/(vertexArray[2][1]-vertexArray[1][1]);

						ddaArray[2].slopeNX = (normalArray[2][0]-normalArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeNY = (normalArray[2][1]-normalArray[1][1])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeNZ = (normalArray[2][2]-normalArray[1][2])/(vertexArray[2][1]-vertexArray[1][1]);

						ddaArray[2].slopeU = (uvArray[2][0]-uvArray[1][0])/(vertexArray[2][1]-vertexArray[1][1]);
						ddaArray[2].slopeV = (uvArray[2][1]-uvArray[1][1])/(vertexArray[2][1]-vertexArray[1][1]);
					}

					deltaY = ceil(vertexArray[0][1]) - vertexArray[0][1];
					deltaY23 = ceil(vertexArray[1][1]) - vertexArray[1][1];
					if(vertexArray[1][1] != vertexArray[0][1]){
						//assign left or right edges
						if(ddaArray[0].slopeX < ddaArray[1].slopeX){
							leftEdge = &(ddaArray[0]);
							rightEdge = &(ddaArray[1]);
						}else{
							leftEdge = &(ddaArray[1]);
							rightEdge = &(ddaArray[0]);
						}

						//advance 1-2 DDA0 current position to top ceilling position
						ddaArray[0].current[0] = ddaArray[0].current[0] + ddaArray[0].slopeX * deltaY;
						ddaArray[0].current[1] = ddaArray[0].current[1] + deltaY;
						ddaArray[0].current[2] = ddaArray[0].current[2] + ddaArray[0].slopeZ * deltaY;

						ddaArray[0].currentNormal[0] = ddaArray[0].currentNormal[0] + ddaArray[0].slopeNX * deltaY;
						ddaArray[0].currentNormal[1] = ddaArray[0].currentNormal[1] + ddaArray[0].slopeNY * deltaY;
						ddaArray[0].currentNormal[2] = ddaArray[0].currentNormal[2] + ddaArray[0].slopeNZ * deltaY;

						ddaArray[0].currentUV[0] = ddaArray[0].currentUV[0] + ddaArray[0].slopeU * deltaY;
						ddaArray[0].currentUV[1] = ddaArray[0].currentUV[1] + ddaArray[0].slopeV * deltaY;

						//advance 1-3 DDA1 current position to top ceilling position
						ddaArray[1].current[0] = ddaArray[1].current[0] + ddaArray[1].slopeX * deltaY;
						ddaArray[1].current[1] = ddaArray[1].current[1] + deltaY;
						ddaArray[1].current[2] = ddaArray[1].current[2] + ddaArray[1].slopeZ * deltaY;

						ddaArray[1].currentNormal[0] = ddaArray[1].currentNormal[0] + ddaArray[1].slopeNX * deltaY;
						ddaArray[1].currentNormal[1] = ddaArray[1].currentNormal[1] + ddaArray[1].slopeNY * deltaY;
						ddaArray[1].currentNormal[2] = ddaArray[1].currentNormal[2] + ddaArray[1].slopeNZ * deltaY;

						ddaArray[1].currentUV[0] = ddaArray[1].currentUV[0] + ddaArray[1].slopeU * deltaY;
						ddaArray[1].currentUV[1] = ddaArray[1].currentUV[1] + ddaArray[1].slopeV * deltaY;

						//advance 2-3 DDA2 current position to top ceilling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY23;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY23;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY23;

						ddaArray[2].currentNormal[0] = ddaArray[2].currentNormal[0] + ddaArray[2].slopeNX * deltaY23;
						ddaArray[2].currentNormal[1] = ddaArray[2].currentNormal[1] + ddaArray[2].slopeNY * deltaY23;
						ddaArray[2].currentNormal[2] = ddaArray[2].currentNormal[2] + ddaArray[2].slopeNZ * deltaY23;

						ddaArray[2].currentUV[0] = ddaArray[2].currentUV[0] + ddaArray[2].slopeU * deltaY23;
						ddaArray[2].currentUV[1] = ddaArray[2].currentUV[1] + ddaArray[2].slopeV * deltaY23;
					}else{
						leftEdge = &(ddaArray[1]);
						rightEdge = &(ddaArray[2]);

						//advance 2-3 DDA2 current position to top ceiling position
						ddaArray[2].current[0] = ddaArray[2].current[0] + ddaArray[2].slopeX * deltaY;
						ddaArray[2].current[1] = ddaArray[2].current[1] + deltaY;
						ddaArray[2].current[2] = ddaArray[2].current[2] + ddaArray[2].slopeZ * deltaY;
					}		

					while(ddaArray[1].current[1] <= vertexArray[2][1] && ddaArray[1].current[1] < 257 ){

						//switch from 1-2 edge to 2-3 edge
						if(ddaArray[0].current[1] >= vertexArray[1][1] && vertexArray[1][1] != vertexArray[0][1]){
							if(ddaArray[0].slopeX < ddaArray[1].slopeX){
								leftEdge = &(ddaArray[2]);
							}else{
								rightEdge = &(ddaArray[2]);
							}
						}

						//inialize the span DDA
						for(j = 0; j < 3; j++){
							spanDDA.start[j] = leftEdge->current[j];
							spanDDA.end[j] = rightEdge->current[j];
							spanDDA.current[j] = leftEdge->current[j];

							spanDDA.startNormal[j] = leftEdge->currentNormal[j];
							spanDDA.endNormal[j] = rightEdge->currentNormal[j];
							spanDDA.currentNormal[j] = leftEdge->currentNormal[j];

							if(j != 2){
								spanDDA.startUV[j] = leftEdge->currentUV[j];
								spanDDA.endUV[j] = rightEdge->currentUV[j];
								spanDDA.currentUV[j] = leftEdge->currentUV[j];
							}
						}

						spanDDA.slopeX = 0.0f;
						spanDDA.slopeZ = (rightEdge->current[2]-leftEdge->current[2])/(rightEdge->current[0]-leftEdge->current[0]);

						spanDDA.slopeNX = (rightEdge->currentNormal[0]-leftEdge->currentNormal[0])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeNY = (rightEdge->currentNormal[1]-leftEdge->currentNormal[1])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeNZ = (rightEdge->currentNormal[2]-leftEdge->currentNormal[2])/(rightEdge->current[0]-leftEdge->current[0]);

						spanDDA.slopeU = (rightEdge->currentUV[0]-leftEdge->currentUV[0])/(rightEdge->current[0]-leftEdge->current[0]);
						spanDDA.slopeV = (rightEdge->currentUV[1]-leftEdge->currentUV[1])/(rightEdge->current[0]-leftEdge->current[0]);

						//advance span DDA current position to left-most covered pixel
						deltaX = ceil(leftEdge->current[0]) - leftEdge->current[0];
						spanDDA.current[0] = spanDDA.current[0] + deltaX;
						spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ * deltaX;
						
						spanDDA.currentNormal[0] = spanDDA.currentNormal[0] + spanDDA.slopeNX * deltaX;
						spanDDA.currentNormal[1] = spanDDA.currentNormal[1] + spanDDA.slopeNY * deltaX;
						spanDDA.currentNormal[2] = spanDDA.currentNormal[2] + spanDDA.slopeNZ * deltaX;

						spanDDA.currentUV[0] = spanDDA.currentUV[0] + spanDDA.slopeU * deltaX;
						spanDDA.currentUV[1] = spanDDA.currentUV[1] + spanDDA.slopeV * deltaX;

						while(spanDDA.current[0] <= spanDDA.end[0]){

							//Phong shading unwarp process: transform UV back to uv
							vzPrime2 = spanDDA.current[2]/((float)INT_MAX - spanDDA.current[2]);
							uTemp = spanDDA.currentUV[0]*(vzPrime2+1.0);
							vTemp = spanDDA.currentUV[1]*(vzPrime2+1.0);
							render->tex_fun(uTemp, vTemp, kColor);
							for(j = 0; j < 3; j++){
								render->Kd[j] = kColor[j];
								render->Ka[j] = kColor[j];
							}

							//write color value into the frame buffer
							x = spanDDA.current[0];
							y = spanDDA.current[1];

							//compute color at each pixel
							for(j = 0; j < 3; j++){
								tempVector1[j] = 0.0;
								tempVector2[j] = 0.0;
								pixelColor[j] = 0.0;
							}
							vectorNormalize(spanDDA.currentNormal);

							for(j = 0; j < render->numlights; j++){
								resultNL = vectorDotProduct(spanDDA.currentNormal, render->lights[j].direction);
								resultNE = vectorDotProduct(spanDDA.currentNormal, eVector);

								if(resultNL * resultNE > 0){
									if(resultNL < 0 && resultNE < 0){
										spanDDA.currentNormal[0] = -1 * spanDDA.currentNormal[0];
										spanDDA.currentNormal[1] = -1 * spanDDA.currentNormal[1];
										spanDDA.currentNormal[2] = -1 * spanDDA.currentNormal[2];
										resultNL = vectorDotProduct(spanDDA.currentNormal, render->lights[j].direction);
										resultNE = vectorDotProduct(spanDDA.currentNormal, eVector);
									}

									vectorScale(2*resultNL, spanDDA.currentNormal);
									vectorConstruct(render->lights[j].direction, spanDDA.currentNormal, rVector);
									vectorScale(1/(2*resultNL), spanDDA.currentNormal);
									resultRE = vectorDotProduct(rVector, eVector) < 0 ? 0 : vectorDotProduct(rVector, eVector);
									result = pow(resultRE, render->spec);
									vectorScale(result, render->lights[j].color, sumVector1);
									vectorAdd(tempVector1, sumVector1);

									vectorScale(resultNL, render->lights[j].color, sumVector2);
									vectorAdd(tempVector2, sumVector2);
								}
							}

							vectorMultiply(tempVector1, render->Ks);
							vectorMultiply(tempVector2, render->Kd);
							vectorMultiply(render->ambientlight.color, render->Ka, tempVector3);

							vectorAdd(pixelColor, tempVector1);
							vectorAdd(pixelColor, tempVector2);
							vectorAdd(pixelColor, tempVector3);

							GzPutDisplay(render->display, x, y, ctoi(pixelColor[0]), ctoi(pixelColor[1]), ctoi(pixelColor[2]),
									1, spanDDA.current[2]);

							//update span DDA current position
							(spanDDA.current[0])++;
							spanDDA.current[2] = spanDDA.current[2] + spanDDA.slopeZ;

							spanDDA.currentNormal[0] = spanDDA.currentNormal[0] + spanDDA.slopeNX;
							spanDDA.currentNormal[1] = spanDDA.currentNormal[1] + spanDDA.slopeNY;
							spanDDA.currentNormal[2] = spanDDA.currentNormal[2] + spanDDA.slopeNZ;

							spanDDA.currentUV[0] = spanDDA.currentUV[0] + spanDDA.slopeU;
							spanDDA.currentUV[1] = spanDDA.currentUV[1] + spanDDA.slopeV;
						}

						//update left and right edge current position
						leftEdge->current[0] = leftEdge->current[0] + leftEdge->slopeX;
						(leftEdge->current[1])++;
						leftEdge->current[2] = leftEdge->current[2] + leftEdge->slopeZ;

						leftEdge->currentNormal[0] = leftEdge->currentNormal[0] + leftEdge->slopeNX;
						leftEdge->currentNormal[1] = leftEdge->currentNormal[1] + leftEdge->slopeNY;
						leftEdge->currentNormal[2] = leftEdge->currentNormal[2] + leftEdge->slopeNZ;

						leftEdge->currentUV[0] = leftEdge->currentUV[0] + leftEdge->slopeU;
						leftEdge->currentUV[1] = leftEdge->currentUV[1] + leftEdge->slopeV;

						rightEdge->current[0] = rightEdge->current[0] + rightEdge->slopeX;
						(rightEdge->current[1])++;
						rightEdge->current[2] = rightEdge->current[2] + rightEdge->slopeZ;

						rightEdge->currentNormal[0] = rightEdge->currentNormal[0] + rightEdge->slopeNX;
						rightEdge->currentNormal[1] = rightEdge->currentNormal[1] + rightEdge->slopeNY;
						rightEdge->currentNormal[2] = rightEdge->currentNormal[2] + rightEdge->slopeNZ;

						rightEdge->currentUV[0] = rightEdge->currentUV[0] + rightEdge->slopeU;
						rightEdge->currentUV[1] = rightEdge->currentUV[1] + rightEdge->slopeV;
					}


		}
	}

	return GZ_SUCCESS;
}

/* NOT part of API - just for general assistance */

__host__ __device__ short	ctoi(float color)		/* convert float color to GzIntensity short */
{
//	if (color > 1.0) color = 1.0;
	return(short)((int)(color * ((1 << 12) - 1)));
}


 __host__ __device__ bool vectorZero(GzCoord vector){

	if(vector[0] == 0 && vector[1] == 0 && vector[2] == 0){
		return true;
	}else{
		return false;
	}
}

 __host__ __device__ void vectorAdd(GzCoord vector1, GzCoord vector2){
	vector1[0] = vector1[0] + vector2[0];
	vector1[1] = vector1[1] + vector2[1];
	vector1[2] = vector1[2] + vector2[2];
}

 __host__ __device__ void vectorAdd(GzCoord vector1, GzCoord vector2, GzCoord vector3){
	vector3[0] = vector1[0] + vector2[0];
	vector3[1] = vector1[1] + vector2[1];
	vector3[2] = vector1[2] + vector2[2];
}

 __host__ __device__ void vectorMultiply(GzCoord vector1, GzCoord vector2){
	vector1[0] = vector1[0] * vector2[0];
	vector1[1] = vector1[1] * vector2[1];
	vector1[2] = vector1[2] * vector2[2];
}

 __host__ __device__ void vectorMultiply(GzCoord vector1, GzCoord vector2, GzCoord vector3){
	vector3[0] = vector1[0] * vector2[0];
	vector3[1] = vector1[1] * vector2[1];
	vector3[2] = vector1[2] * vector2[2];
}

 __host__ __device__ void vectorConstruct(GzCoord vector1, GzCoord vector2, GzCoord vector){
	vector[0] = vector2[0] - vector1[0];
	vector[1] = vector2[1] - vector1[1];
	vector[2] = vector2[2] - vector1[2];
}

 __host__ __device__ void vectorNormalize(GzCoord vector){
	float length = sqrt(vector[0]*vector[0] + vector[1]*vector[1] + vector[2]*vector[2]);
	vector[0] = vector[0]/length;
	vector[1] = vector[1]/length;
	vector[2] = vector[2]/length;
}

 __host__ __device__ float vectorDotProduct(GzCoord vector1, GzCoord vector2){
	return vector1[0]*vector2[0] + vector1[1]*vector2[1] + vector1[2]*vector2[2];
}

 __host__ __device__ void vectorCrossProduct(GzCoord vector1, GzCoord vector2, GzCoord product){
	product[0] = vector1[1]*vector2[2] - vector1[2]*vector2[1];
	product[1] = vector1[2]*vector2[0] - vector1[0]*vector2[2];
	product[2] = vector1[0]*vector2[1] - vector1[1]*vector2[0];
}

 __host__ __device__ void vectorScale(float scale, GzCoord vector){
	vector[0] = vector[0] * scale;
	vector[1] = vector[1] * scale;
	vector[2] = vector[2] * scale;
}

 __host__ __device__ void vectorScale(float scale, GzCoord vector1, GzCoord vector2){
	vector2[0] = vector1[0] * scale;
	vector2[1] = vector1[1] * scale;
	vector2[2] = vector1[2] * scale;
}

 __host__ __device__ float vectorTransform(GzCoord vector, float w, GzMatrix matrix){
	float vector0 = matrix[0][0]*vector[0] + matrix[0][1]*vector[1] + matrix[0][2]*vector[2] + matrix[0][3]*w;
	float vector1 = matrix[1][0]*vector[0] + matrix[1][1]*vector[1] + matrix[1][2]*vector[2] + matrix[1][3]*w;
	float vector2 = matrix[2][0]*vector[0] + matrix[2][1]*vector[1] + matrix[2][2]*vector[2] + matrix[2][3]*w;
	float temp = matrix[3][0]*vector[0] + matrix[3][1]*vector[1] + matrix[3][2]*vector[2] + matrix[3][3]*w;
	vector[0] = vector0;
	vector[1] = vector1;
	vector[2] = vector2;
	return temp;
}

 __host__ __device__ void matrixMultiply(GzMatrix matrixA, GzMatrix matrixB, GzMatrix matrixC){
	int i, j, k;
	float temp;

	for(i = 0; i < 4; i++){
		for(j = 0; j < 4; j++){
			temp = 0.0;
			for(k = 0; k < 4; k++){
				temp+=matrixA[i][k]*matrixB[k][j];
			}
			matrixC[i][j] = temp;
		}
	}
}

 __host__ __device__ float vectorLength(GzCoord vector){

	 return sqrtf(vector[0]*vector[0] + vector[1]*vector[1] + vector[2]*vector[2]); 
}
